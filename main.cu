#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

// Write down the kernels here

/*Initially, I calculate the prefix sum of the facility capacities using the reduction method to facilitate the mapping of center numbers and facility IDs for processing requests. Then, I create threads that correspond to the total number of facilities available. Each thread is assigned to represent a specific facility.*/

/*Subsequently, I create a worklist to accommodate all requests that will utilize a particular facility. I also initialize a slot array that has 25 hours and set all its elements to zero. After that, I check if a facility is available for a specific request, and if it is, I proceed with processing that request. If not, I leave the facility as it is without updating anything.*/

__global__ void  exclusivesum(int n,int *garr , int* ccopy,int *csum) // to calculate exclusive sum of array, calling as kernel
{
    int id=threadIdx.x+ blockDim.x*blockIdx.x;
    int th = threadIdx.x;
    int blc = blockDim.x*blockIdx.x;
   
    if((th + blc)<n)
    {
      int aa = ccopy[th + blc];
      garr[id]= garr[id] - aa;
    }
    if(id < 0)
      return;

     if((threadIdx.x+ blockDim.x*blockIdx.x) == n-1)
      {
        int temp = ccopy[n-1] + garr[n-1];
        csum[0] = temp;
      }
}
__global__ void  dkernel(int n,int *a,int start, int end)
{     
     int th = threadIdx.x;
     
     int id=start+th;
     
     int N,M;
     M=end-start;
     N = M+1;
     if(id >= n)
     {
      return;
     }
  
     if((start+th)<n)
     {
          int  tmp;
          
          
        int off = 1;
        while(off < (M+1))
        {
                
            if(th>=off)
            { 
              int tp = id - off;
                tmp=a[tp];
            }
    
            __syncthreads();
            if(start < 0 && N == th)
            return;
 
           if(th>=off || th < 0)
              a[start+threadIdx.x]= a[id] + tmp;
   
           __syncthreads();
           off = off * 2;
      
        }
          
      
    }
  
}


__global__ void  dke(int *sum, int *gcentre,int *gfacility,int *gcapacity,int *gfac_ids,int *gsucc_reqs,int *gtot_reqs,int *greq_id,int *greq_cen,int *greq_fac,int *greq_start,int *greq_slots,int *gfacps,int R,int N)
{
    if(!(threadIdx.x+1))
    {
      return;
    }
    int thd = threadIdx.x;
    int id=threadIdx.x + blockIdx.x * blockDim.x;
    int blk = blockIdx.x;
    if(id >= sum[0])
    {
      return;
    }
    
    if((blk * blockDim.x + thd)< sum[0])
    {
     
    
    int slot[25];
    int worklist[BLOCKSIZE];
    
    
   
    for(int ii = 0; ii < 25; ii++)
    slot[ii] = 0;
    
    
    
    int centerno=-1,facilityno=-1;
    for(int i = 0; i < N; i++)
    {
      int facid = gfacps[i];
      if( i < slot[0])
      return;
      if(id < facid)
        {
          if(id < sum[0])
          {centerno = i - 1;
          int gg = gfacps[i - 1];
          facilityno =  (blk * blockDim.x + thd) - gg;
          break;}
        }
    }

    if(id < sum[0] && centerno==-1)
     {
        int lastInd = N - 1;
        int aai = gfacps[lastInd];
        centerno=N-1;
        facilityno = id - aai;
     }
     int counter  = 0;
    for(int ii = 0; ii < BLOCKSIZE; ii++)
    worklist[ii] = BLOCKSIZE + 1;
    int j = 0;
    while(j < R)
    {
      int cen_req = greq_cen[j];
      int fac_req = greq_fac[j];
      if(cen_req == centerno && fac_req == facilityno)
      {
        if(greq_fac[j] == facilityno)
          {
            worklist[counter] = greq_id[j];
            counter++;
          }
      }
      j = j+1;
    }
    int nor = counter;
    

    for(int i=0;i<nor;i++)
    {
        int var = 0;
        int check= -1;
        for(int p=greq_start[worklist[i]];p<greq_start[worklist[i]] + greq_slots[worklist[i]];p++)
        {
           if(slot[p]>=gcapacity[id])
            { 
              check=1;
              break;
            } 
        }

        if(check == -1)
        {
          int k = greq_start[worklist[i]];
          while(k < greq_start[worklist[i]] + greq_slots[worklist[i]])
            {
               if(check == -1)
               {
                slot[k]++;
               }
               k++;
            }
           
           atomicAdd(&gsucc_reqs[centerno],1);

        }
        
        var = var + check;

    }
    }

    
}
//***********************************************
int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    int k1=0 , k2 = 0;
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[k1] );
        k1++;
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[k2]);
        k2++;     
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		


    //*********************************
    // Call the kernels here
    
    int *csum;
    int *greq_id, *greq_cen,*gcapacity;
    int *gfac_ids, *greq_fac, *greq_start, *greq_slots;
    hipMalloc(&greq_fac,sizeof(int)*R);

    
    int *gcentre,*gfacility, *gsucc_reqs, *gtot_reqs;
    
    hipMalloc(&csum,sizeof(int));
    hipMalloc(&greq_id,sizeof(int)*R);
    hipMalloc(&greq_cen,sizeof(int)*R);
    hipMalloc(&greq_start,sizeof(int)*R);
    hipMemcpy(greq_fac, req_fac, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_start, req_start, R * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&gsucc_reqs,sizeof(int)*N);
    hipMemcpy(greq_id, req_id, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(greq_cen, req_cen, R * sizeof(int), hipMemcpyHostToDevice);
    
    hipMalloc(&greq_slots,sizeof(int)*R);
    hipMalloc(&gcentre,sizeof(int)*N);
    
    
    
    hipMemcpy(greq_slots, req_slots, R * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gcentre,centre , N* sizeof(int), hipMemcpyHostToDevice);
    
   
    
    
    hipMemcpy(gsucc_reqs,succ_reqs ,  N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&gfacility,sizeof(int)*N);
    hipMalloc(&gcapacity,sizeof(int)*max_P * N );
    
    hipMalloc(&gfac_ids,sizeof(int)*max_P * N );
    
    int *gfacps,*ccopy;
    hipMalloc(&gfacps,sizeof(int)*(N));

    hipMalloc(&gtot_reqs,sizeof(int)*N);
    hipMemcpy(gfacility, facility, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gcapacity,capacity , max_P * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(gsucc_reqs, 0, N * sizeof(int));
    hipMemcpy(gfac_ids, fac_ids, max_P * N * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&ccopy,sizeof(int)*(N));
    
    hipMemcpy(gtot_reqs,tot_reqs, N * sizeof(int), hipMemcpyHostToDevice);

    
    
    
    
    hipMemcpy(ccopy,facility, N * sizeof(int), hipMemcpyHostToDevice);
    
    int noofblocks= ceil((float)N/1023);
    hipMemcpy(gfacps,facility, N * sizeof(int), hipMemcpyHostToDevice);

    int sum[1];
    int i = 0;
    while(i < noofblocks)
    {  
        int start=i*1023;
        int end;
        
        if(i < noofblocks)
        {
        int nn = noofblocks -1;
        if(i==nn)
         end=N-1;
        else
         end= (i+1)*1022;
        }
       
       if(i==0)
       dkernel<<<1,1023>>>(N,gfacps,start,end);
       else
       dkernel<<<1,1023+1>>>(N,gfacps,start-1,end);
       hipDeviceSynchronize();
       i++;
    }
    if(N)
    {
      exclusivesum<<<noofblocks,BLOCKSIZE>>>(N,gfacps,ccopy,csum);
      hipDeviceSynchronize();
    }
    hipMemcpy(sum,csum,sizeof(int),hipMemcpyDeviceToHost);

    
    int numBlocks = (sum[0] + BLOCKSIZE - 1);
    int final = numBlocks/BLOCKSIZE;
    
    dke<<<final, BLOCKSIZE>>>(csum, gcentre, gfacility, gcapacity, gfac_ids, gsucc_reqs, gtot_reqs, greq_id, greq_cen, greq_fac, greq_start, greq_slots, gfacps, R, N);
    hipFree(gcentre);
    hipFree(gfacility);
    hipFree(gcapacity);
    hipMemcpy(sum,csum,sizeof(int),hipMemcpyDeviceToHost);

    if(sum[0] < 0)
    {
      sum[0] = success;
    }
    hipMemcpy(succ_reqs ,gsucc_reqs,  N * sizeof(int), hipMemcpyDeviceToHost);

    
    int iii = 0;
    while(iii<N)
     {
        int a = iii++;
        success = success+ succ_reqs[a];

     }
    int ff;
    if(success + 1)
    
    {
      ff = R - success + sum[0];
    } 
    fail = ff - sum[0];

    //********************************
    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    //printf("%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
        //printf("%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}
